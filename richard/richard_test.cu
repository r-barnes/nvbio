#include "hip/hip_runtime.h"
#include <nvbio-test/alignment_test_utils.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/cuda/ldg.h>
#include <nvbio/basic/cached_iterator.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/packedstream_loader.h>
#include <nvbio/basic/vector_view.h>
#include <nvbio/basic/vector.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/basic/dna.h>
#include <nvbio/alignment/alignment.h>
#include <nvbio/alignment/batched.h>
#include <nvbio/alignment/sink.h>
#include <thrust/device_vector.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>

using namespace nvbio;

using namespace nvbio::aln;

enum { CACHE_SIZE = 32 };
typedef nvbio::lmem_cache_tag<CACHE_SIZE>                                       lmem_cache_tag_type;
typedef nvbio::uncached_tag                                                     uncached_tag_type;

//
// An alignment stream class to be used in conjunction with the BatchAlignmentScore class
//
template <typename t_aligner_type, uint32 M, uint32 N, typename cache_type = lmem_cache_tag_type>
struct AlignmentStream
{
    typedef t_aligner_type                                                          aligner_type;

    typedef nvbio::cuda::ldg_pointer<uint32>                                        storage_iterator;

    typedef nvbio::PackedStringLoader<storage_iterator,4,false,cache_type>          pattern_loader_type;
    typedef typename pattern_loader_type::input_iterator                            uncached_pattern_iterator;
    typedef typename pattern_loader_type::iterator                                  pattern_iterator;
    typedef nvbio::vector_view<pattern_iterator>                                    pattern_string;

    typedef nvbio::PackedStringLoader<storage_iterator,2,false,cache_type>          text_loader_type;
    typedef typename text_loader_type::input_iterator                               uncached_text_iterator;
    typedef typename text_loader_type::iterator                                     text_iterator;
    typedef nvbio::vector_view<text_iterator>                                       text_string;

    // an alignment context
    struct context_type
    {
        int32                   min_score;
        aln::BestSink<int32>    sink;
    };
    // a container for the strings to be aligned
    struct strings_type
    {
        pattern_loader_type     pattern_loader;
        text_loader_type        text_loader;
        pattern_string          pattern;
        trivial_quality_string  quals;
        text_string             text;
    };

    // constructor
    AlignmentStream(
        aligner_type        _aligner,
        const uint32        _count,
        const uint32*       _patterns,
        const uint32*       _text,
               int16*       _scores) :
        m_aligner( _aligner ), m_count(_count), m_patterns(storage_iterator(_patterns)), m_text(storage_iterator(_text)), m_scores(_scores) {}

    // get the aligner
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    const aligner_type& aligner() const { return m_aligner; };

    // return the maximum pattern length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 max_pattern_length() const { return M; }

    // return the maximum text length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 max_text_length() const { return N; }

    // return the stream size
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 size() const { return m_count; }

    // return the i-th pattern's length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 pattern_length(const uint32 i, context_type* context) const { return M; }

    // return the i-th text's length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 text_length(const uint32 i, context_type* context) const { return N; }

    // initialize the i-th context
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    bool init_context(
        const uint32    i,
        context_type*   context) const
    {
        context->min_score = Field_traits<int32>::min();
        return true;
    }

    // initialize the i-th context
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    void load_strings(
        const uint32        i,
        const uint32        window_begin,
        const uint32        window_end,
        const context_type* context,
              strings_type* strings) const
    {
        strings->pattern = pattern_string( M,
            strings->pattern_loader.load(
                m_patterns + i * M,
                M,
                make_uint2( window_begin, window_end ),
                false ) );

        strings->text = text_string( N, strings->text_loader.load( m_text + i * N, N ) );
    }

    // handle the output
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    void output(
        const uint32        i,
        const context_type* context) const
    {
        // copy the output score
        m_scores[i] = context->sink.score;
    }

    aligner_type                m_aligner;
    uint32                      m_count;
    uncached_pattern_iterator   m_patterns;
    uncached_text_iterator      m_text;
    int16*                      m_scores;
};

//
// A class for making a single alignment test, testing both scoring and traceback
//
struct SingleTest
{
    thrust::host_vector<uint8>   str_hvec;
    thrust::host_vector<uint8>   ref_hvec;
    thrust::device_vector<uint8> str_dvec;
    thrust::device_vector<uint8> ref_dvec;
    thrust::device_vector<float> temp_dvec;
    thrust::device_vector<float> score_dvec;
    thrust::device_vector<uint2> sink_dvec;

    // test banded alignment
    //
    // \param test              test name
    // \param aligner           alignment algorithm
    // \param ref_alignment     reference alignment string
    //
    template <uint32 BLOCKDIM, uint32 BAND_LEN, const uint32 N, const uint32 M, typename aligner_type>
    void banded(const char* test, const aligner_type aligner, const char* ref_alignment)
    {
        NVBIO_VAR_UNUSED const uint32 CHECKPOINTS = 32u;

        const uint8* str_hptr = nvbio::raw_pointer( str_hvec );
        const uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );

        const int32 ref_score = ref_banded_sw<M,N,BAND_LEN>( str_hptr, ref_hptr, 0u, aligner );

        aln::BestSink<int32> sink;
        aln::banded_alignment_score<BAND_LEN>(
            aligner,
            vector_view<const uint8*>( M, str_hptr ),
            trivial_quality_string(),
            vector_view<const uint8*>( N, ref_hptr ),
            -1000,
            sink );

        const int32 cpu_score = sink.score;
        if (cpu_score != ref_score)
        {
            log_error(stderr, "    expected %s score %d, got: %d\n", test, ref_score, cpu_score);
            exit(1);
        }

        TestBacktracker backtracker;
        backtracker.clear();

        const Alignment<int32> aln = aln::banded_alignment_traceback<BAND_LEN,1024u,CHECKPOINTS>(
            aligner,
            vector_view<const uint8*>( M, str_hptr ),
            trivial_quality_string(),
            vector_view<const uint8*>( N, ref_hptr ),
            -1000,
            backtracker );

        const int32 aln_score = backtracker.score( aligner, aln.source.x, str_hptr, ref_hptr );
        const std::string aln_string = rle( backtracker.aln ).c_str();
        if (aln_score != ref_score)
        {
            log_error(stderr, "    expected %s backtracking score %d, got %d\n", ref_score, aln_score);
            log_error(stderr, "    %s - %d - [%u, %u] x [%u, %u]\n", aln_string.c_str(), aln.score, aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
            exit(1);
        }
        fprintf(stderr, "    %15s : ", test);
        fprintf(stderr, "%d - %s - [%u:%u] x [%u:%u]\n", aln.score, aln_string.c_str(), aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
        if (strcmp( ref_alignment, aln_string.c_str() ) != 0)
        {
            log_error(stderr, "    expected %s, got %s\n", ref_alignment, aln_string.c_str());
            exit(1);
        }
    }
};

// execute and time the batch_banded_score<scheduler> algorithm for all possible schedulers
//
template <uint32 BAND_LEN, uint32 N, uint32 M, typename aligner_type>
void batch_banded_score_profile_all(
    const aligner_type              aligner,
    const uint32                    n_tasks,
    thrust::device_vector<uint32>&  pattern_dvec,
    thrust::device_vector<uint32>&  text_dvec,
    thrust::device_vector<int16>&   score_dvec)
{
    // create a stream
    typedef AlignmentStream<aligner_type,M,N> stream_type;
    stream_type stream(
        aligner,
        n_tasks,
        nvbio::raw_pointer( pattern_dvec ),
        nvbio::raw_pointer( text_dvec ),
        nvbio::raw_pointer( score_dvec ) );

    // setup a batch
    //TODO: Can also use: DeviceStagedThreadScheduler. Maybe also DeviceWarpScheduler?
    typedef aln::BatchedBandedAlignmentScore<BAND_LEN,stream_type, DeviceThreadScheduler> batch_type;
    batch_type batch;

    // alloc all the needed temporary storage
    const uint64 temp_size = batch_type::max_temp_storage(
        stream.max_pattern_length(),
        stream.max_text_length(),
        stream.size() );

    thrust::device_vector<uint8> temp_dvec( temp_size );

    Timer timer;
    timer.start();
    batch.enact( stream, temp_size, nvbio::raw_pointer( temp_dvec ) );
    hipDeviceSynchronize();
    timer.stop();

    const float time = timer.seconds();

    fprintf(stderr,"  %5.1f", 1.0e-9f * float(n_tasks*uint64(BAND_LEN*M))*(1/time) );
    fprintf(stderr, " GCUPS\n");
}



int main(int argc, char* argv[])
{
    NVBIO_VAR_UNUSED uint32 N_WARP_TASKS     = 4096;
                     uint32 N_THREAD_TASKS   = 128*1024;

    fprintf(stderr,"testing alignment... started\n");

    const uint32 BAND_LEN = 15u;
    const uint32 N_TASKS  = N_THREAD_TASKS;
    const uint32 M = 150;
    const uint32 N = M+BAND_LEN;

    const uint32 M_WORDS = (M + 7)  >> 3;
    const uint32 N_WORDS = (N + 15) >> 4;

    thrust::host_vector<uint32> str( M_WORDS * N_TASKS );
    thrust::host_vector<uint32> ref( N_WORDS * N_TASKS );

    LCG_random rand;
    fill_packed_stream<4u>( rand, 4u, M * N_TASKS, nvbio::raw_pointer( str ) );
    fill_packed_stream<2u>( rand, 4u, N * N_TASKS, nvbio::raw_pointer( ref ) );

    thrust::device_vector<uint32> str_dvec( str );
    thrust::device_vector<uint32> ref_dvec( ref );
    thrust::device_vector<int16>  score_dvec( N_TASKS );

    fprintf(stderr,"  testing banded Smith-Waterman scoring speed...\n");
    //Also aln::SEMI_GLOBAL, aln::GLOBAL
    fprintf(stderr,"    %15s : ", "local");
    batch_banded_score_profile_all<BAND_LEN,N,M>(
        make_smith_waterman_aligner<aln::LOCAL>( aln::SimpleSmithWatermanScheme(2,-1,-1,-1) ),
        N_TASKS,
        str_dvec,
        ref_dvec,
        score_dvec
    );

    return 0;
}
